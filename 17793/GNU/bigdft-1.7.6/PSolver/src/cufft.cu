#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
 
#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
extern "C" void synchronize_() {
 
  hipDeviceSynchronize();
}

// allocate device memory
extern "C" void cudamalloc_(int *size, Real **d_data,int *ierr) {

  *ierr = hipMalloc((void**)d_data, sizeof(Real)*(*size));
  if( hipGetLastError() != hipSuccess)
      printf("allocate error\n");
}

extern "C" void cudafree_(Real **d_data) {

  hipFree(*d_data);
}

extern "C" void cufftdestroy_(hipfftHandle *plan) {

  hipfftDestroy(*plan);
}

// set device memory
extern "C" void FC_FUNC_(reset_gpu_data, RESET_GPU_DATA)(int *size, Real* h_data, Real **d_data){

 hipMemcpy(*d_data, h_data, sizeof(Real)*(*size),
         hipMemcpyHostToDevice);
 if( hipGetLastError() != hipSuccess)
      printf("transfer error\n");

}

// read device memory
extern "C" void FC_FUNC_(get_gpu_data, GET_GPU_DATA)(int *size, Real *h_data, Real **d_data) {

 hipMemcpy(h_data, *d_data, sizeof(Real)*(*size),
         hipMemcpyDeviceToHost);
 if (hipGetLastError() != hipSuccess)
        printf("transfer back error\n");
}


// transpose
__global__ void transpose(Complex *idata, Complex *odata,
        int width, int height)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}

// transpose together with spread operation
__global__ void transpose_spread(Complex *idata, Complex *odata, 
	int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_out / bign_h;
  int mod = index_out % bign_h;
  index_out = div * (bign_h << 1) + mod+bign_h;
  int plus = -bign_h;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
    #ifdef DOUBLE
      odata[index_out + plus] = make_double2(0., 0.);
    #else
      odata[index_out + plus] = make_float2(0.f, 0.f);
    #endif
  }
}

// transpose together with inverse spread operation
__global__ void transpose_spread_i(Complex *idata, Complex *odata,
        int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_in / bign_h;
  int mod = index_in % bign_h;
  index_in = div * (bign_h << 1) + mod;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width)
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
}

// spread operation
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;
 
   Real res = (tid >= spitch) ? src[bid * spitch + tid-spitch] : 0.0;
   if( tid < dpitch) {
	dst[bid * dpitch + tid] = res;
   }
}

// inverse spread operation
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;

   Real res = src[bid * dpitch + tid];
   if( tid < dpitch) dst[bid * spitch + tid] = res;
}

// spread operation for 2nd dim
__global__ void spread_y(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = make_double2(0., 0.);
#else
   dst[tid1] = make_float2(0.f, 0.f);
#endif
}

__global__ void spread_y_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = 0.;
#else
   dst[tid1] = 0.f;
#endif
}

__global__ void spread_z(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   //unsigned int tid1 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   src[tid + (gridDim.y * gridDim.x) * blockDim.x] = res;
#ifdef DOUBLE
   src[tid] = 0.0;
#else
   src[tid] = 0.f;
#endif
}


// inverse spread operation for 2nd dim
__global__ void spread_y_i(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid1];
   dst[tid] = res;
}

__global__ void spread_y_i_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid1];
   dst[tid] = res;
}


// multiply with potential
__global__ void multiply_kernel(int nx, int ny, int nz, Complex *d_data, Real *d_kernel, Real scal) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= d_kernel[jj + tj+ k*td]*scal;
     d_data[jj + tj+ k*td].y *= d_kernel[jj + tj+ k*td]*scal;
 }

}

// scale
__global__ void scale_kernel(int nx, int ny, int nz, Complex *d_data, Real mult) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= mult;
     d_data[jj + tj+ k*td].y *= mult;
 }

}

__global__ void zero(int nx, int ny, int nz, Real *z) {

        int tj = threadIdx.x;
        int td = blockDim.x;

	int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

        int jj = ((blockIdx.y)*gridDim.x + (blockIdx.x))*blockData;

        for (int k=0; k<blockData/td; k++) {
        	z[jj + tj+ k*td] = 0.0;
        }
}

__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {

	int tj = threadIdx.x;
        int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y+nz/2)*nx*ny + (blockIdx.x+ny/2)*nx);


        out[jj1+tj+td] = in[jj+tj];

}

__global__ void copy(int nx,int ny,int nz, Real *in, Real *out) {

        int tj = threadIdx.x;
        //int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y)*nx*ny + (blockIdx.x)*nx);

        out[jj+tj] = in[jj1+tj];
}

/************ 1D transform *************/

extern "C" void cuda_1d_plan_(int *NX_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, 1, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 //hipfftPlan1d(plan, NX, Transform, Nbatch );

}

extern "C" void cuda_1d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D forward transform\n");
   }

}

extern "C" void cuda_1d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D inverse transform\n");
   }

}

/************ 2D transform *************/

extern "C" void cuda_2d_plan_(int *NX_p, int *NY_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, NY, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX*NY,
              NULL, 1, NX*NY, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}

extern "C" void cuda_2d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D forward transform\n");
   }

}

extern "C" void cuda_2d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D inverse transform\n");
   }

}

/************ 3D transform *************/

extern "C" void cuda_3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, Transform, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");
}

extern "C" void cuda_3d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D forward transform\n");
   }

}

extern "C" void cuda_3d_inverse_(int *NX_p, int *NY_p, int *NZ_p ,hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D inverse transform\n");
   }

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);

   scale_kernel <<< nBlocks, nThreads >>> (NX,NY,NZ,*d_data2, 1.0/double(NX*NY*NZ));
}

/************ 3D Poisson Solver for periodic boundary *************/

extern "C" void cuda_3d_psolver_cufft3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan, hipfftHandle *plan1) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan1, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}


extern "C" void cuda_3d_psolver_cufft3d_(int *NX_p, int *NY_p, int *NZ_p,hipfftHandle *plan,
             hipfftHandle *plan1, Complex **d_data, Complex **d_data2, Real **d_kernel, Real *scal_p,
	     int *geo1_p, int *geo2_p, int *geo3_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int geo1 = *geo1_p;
 int geo2 = *geo2_p;
 int geo3 = *geo3_p;

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 Real scal = *scal_p;

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NY,NZ,1);

 // copy kernel paramters
 int nthreads = NX/2;
 dim3 nblocks(NY/2,NZ/2,1);

 // spread kernel parameters
 dim3 nblocks_s(zsize,ysize,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

   if (geo1==0 && geo2==0 && geo3==0) {
    src = *d_data;
    dst = *d_data2;
    zero <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst);
    copy_0 <<< nblocks, nthreads  >>> (NX,NY,NZ, (Real*)src, (Real*)dst);
   }
   else {
    if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks_s, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
    }
    if (geo2==0) {
      if (geo1==0) {
        src = *d_data2;
        dst = *d_data;
      } else {
        src = *d_data;
        dst = *d_data2;
      }
      nblocks_s.x=ysize;
      nblocks_s.y=zsize;
      spread_y_r<<<nblocks_s, NX>>>((Real*)src, (Real*)dst);
    }
    if (geo3==0) {
      nblocks_s.x=NY;
      nblocks_s.y=zsize;
      spread_z<<<nblocks_s, NX>>>((Real*)dst, (Real*)src);
    }
   }

   // Forward FFT

   if( hipfftExecD2Z(*plan, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSper forward transform\n");
   }

   // multiply with kernel

   multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

   // Inverse FFT

   if( hipfftExecZ2D(*plan1, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSper inverse transform\n");
   }

   if (geo1==0 && geo2==0 && geo3==0)
     copy <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst, (Real*)src);
   else { 
    if (geo2==0) {
       nblocks_s.x=ysize;
       nblocks_s.y=zsize;
       spread_y_i_r<<<nblocks_s, NX>>>((Real*)dst, (Real*)src);
    }
    if (geo1==0) {
       if (geo2==0) {
        Complex* tmp = src;
        src = dst;
        dst = tmp;
       }
      nblocks_s.x=zsize;
      nblocks_s.y=ysize; 
      spread_i<<<nblocks_s, NX/2>>>((Real*)dst, NX/2, (Real*)src, NX);
    }
   }
}

/************ 3D Poisson Solver for general boundary *************/

extern "C" void FC_FUNC_(cuda_3d_psolver_general_plan, CUDA_3D_PSOLVER_GENERAL_PLAN)(int *N,
                 hipfftHandle *plan, int *switch_alg,
		 int *geo) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 //int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int n1d[3]= {1, 1, 1};

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 n1d[0] = NX;
 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan+1,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NY;
 if(hipfftPlanMany(plan+2,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NZ;
 if(hipfftPlanMany(plan+3,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 *switch_alg = 0;

 /*int nPrimeSize = 17;
 int primeSize[] = {92,104,116,124,136,148,152,164,172,184,188,204,208,220,228,232,248};

 for (int p=0; p<nPrimeSize; p++)
   if (NZ == primeSize[p]) {
     *switch_alg = 1;
     break;
   }

 n1d[0] = NZ;

 int inembed[1];
 int onembed[1];
 inembed[0] = 1;
 onembed[0] = 1;
 if(hipfftPlanMany(plan+4,  1, n1d,
              inembed, NY, 1,
              onembed, NY, 1, Transform, NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");*/

}

extern "C" void FC_FUNC_(cuda_3d_psolver_general, CUDA_3D_PSOLVER_GENERAL)(int *N,
          hipfftHandle *plan,
          Complex **d_data, Complex **d_data2, Real **d_kernel, int *switch_alg,
          int *geo, Real *scal_p) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   if( hipfftExecD2Z(plan[0], (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 1\n");
   }

   if (geo2==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize);
   }

   // Y transform
   if( TransformExec(plan[2], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 2\n");
   }

  // Z transform, on entire cube
  if (!(*switch_alg)) {
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   if( TransformExec(plan[3], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
   }
  }
  else {
   if (geo3==0) {
      nblocks.x=zsize;
      nblocks.y=NX;
      spread_y<<<nblocks, NY>>>(src, dst);
   }

   for(int k=0; k<NX; ++k){
     if( TransformExec(plan[4], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;
  }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
  if (!(*switch_alg)) {
   if( TransformExec(plan[3], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 1\n");
   }

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads >>>(dst,src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads >>>(dst, src,NZ*(NX/2+1),NY);
   }

  }
  else {

   for(int k=0; k<NX; ++k){
     if( TransformExec(plan[4], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;

   if (geo3==0)
      spread_y_i<<<nblocks, NY>>>(dst, src);
  }

  // Y transform

   if( TransformExec(plan[2], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 2\n");
   }

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   if( hipfftExecZ2D(plan[1], src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
   }

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2>>>((Real*)dst, NX/2, (Real*)src, NX);
   }
}


extern "C" void FC_FUNC_(cuda_3d_psolver_plangeneral, CUDA_3D_PSOLVER_PLANGENERAL)(int *N,
          Complex **d_data, Complex **d_data2, Real **d_kernel,
          int *geo, Real *scal_p) {

 hipfftHandle plan;

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 int n1d[3]= {1, 1, 1};

 n1d[0] = NX;
 if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 1\n");

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   if( hipfftExecD2Z(plan, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 1\n");
   }

   if (geo2==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize);
   }

   hipfftDestroy(plan);

   n1d[0] = NY;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 2\n");

   // Y transform
   if( TransformExec(plan, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 2\n");
   }

  // Z transform, on entire cube
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   hipfftDestroy(plan);
   n1d[0] = NZ;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY) != HIPFFT_SUCCESS)
      printf("Error creating plan 3\n");

   if( TransformExec(plan, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
   }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
   if( TransformExec(plan, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 1\n");
   }

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads >>>(dst,src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads >>>(dst, src,NZ*(NX/2+1),NY);
   }

  // Y transform

   hipfftDestroy(plan);
   n1d[0] = NY;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 4\n");

   if( TransformExec(plan, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 2\n");
   }

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   hipfftDestroy(plan);
   n1d[0] = NX;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 5\n");

   if( hipfftExecZ2D(plan, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
   }

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2>>>((Real*)dst, NX/2, (Real*)src, NX);
   }

   hipfftDestroy(plan);
}
