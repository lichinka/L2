/**
 * @file cudafct.cu
 * @author Matthieu Ospici
 * 
 * @brief
 * cudafct is wrapping cuda function calls. This wrapping adds error checking when
 * calling these cuda functions.
 * 
 * @section LICENSE
 * 
 * Copyright (C) 2010 BULL LIG CEA-INAC UJF
 *
 * This file is part of S_GPU library.
 * 
 * S_GPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * S_GPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with S_GPU.  If not, see <http://www.gnu.org/licenses/>.
 */

#ifdef HAVE_CONFIG
#include <config.h>
#endif

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUERR(fct_name) { hipError_t err; \
 if ((err = hipGetLastError()) != hipSuccess) { \
 printf("CUDA error: %s, in file %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); } \
 printf ( "**** ERROR *** : %s\n", fct_name );}


extern "C"
int c_cudaGetDeviceCount ( int *s_gpuCount )
{
	return hipGetDeviceCount ( s_gpuCount );
}

extern "C"
int c_cudaSetDevice ( int device )
{
	return hipSetDevice ( device );
}

extern "C"
int c_cudaMalloc ( void** p, unsigned int memsize )
{
	int ret = hipMalloc ( p, memsize );
	if ( ret != hipSuccess )
	{
		CUERR("c_cuda_malloc");
	}
	return ret;
}

extern "C"
int c_cudaMallocHost ( void** p, unsigned int memsize )
{
	int ret = hipHostMalloc ( p, memsize , hipHostMallocDefault);
	if ( ret != hipSuccess )
	{
		CUERR("c_hipHostMalloc");
	}
	return ret;
}

//run with fstrict-aliasing
typedef union conv_u
{
	float **p;
	void **v;
} conv_u_t;

extern "C"
int c_cudaMallocHostFloat ( float **p, unsigned int memsize )
{
	conv_u_t tmp;
	tmp.p = p;
	return hipHostMalloc ( tmp.v, memsize , hipHostMallocDefault);
}

extern "C"
int c_cuda_get_device ( int *dev )
{
	return hipGetDevice ( dev );
}

extern "C"
int c_cuda_gpu_send_pi ( void *dest, const void *src,  size_t memByte )
{
	// std::cout << "cudaSucess : " << cudaSuccess
	//	    << "cudaErrorInvalidValue : " << cudaErrorInvalidValue
	//	    << "cudaErrorInvalidDevicePointer : " << cudaErrorInvalidDevicePointer
	//	    << "cudaErrorInvalidMemcpyDirection : " << cudaErrorInvalidMemcpyDirection<< std::endl;
	int ret =  hipMemcpyAsync ( dest, src, memByte, hipMemcpyHostToDevice, 0 );
	if ( ret != hipSuccess )
	{
		CUERR("c_cuda_gpu_send_pi");
	}
  // the asynchronous copy is synchronized here
	if ( hipStreamSynchronize ( 0 ) != hipSuccess )
	{
		CUERR("c_cuda_gpu_send_pi stream");
	}
	return ret;
}

extern "C"
int c_cuda_gpu_recv_pi ( void *dest, const void *src,  size_t memByte )
{
	int ret = hipMemcpyAsync ( dest, src, memByte, hipMemcpyDeviceToHost, 0 );
	if ( ret != hipSuccess )
	{
		CUERR("c_cuda_gpu_recv_pi");
	}
	if ( hipStreamSynchronize ( 0 ) != hipSuccess )
	{
		CUERR("c_cuda_gpu_recv_pi stream");
	}
	return ret;
}

extern "C"
int c_cuda_setdevice ( int device )
{
	return hipSetDevice ( device );
}

/*int c_cuda_setdevice_ctx(int device)
{

  CUcontext pCtx;

  CUdevice dev;
  if( cuDeviceGet(&dev, device) != CUDA_SUCCESS)
    {
      CUERR;
      std::cout << "**** ERROR *** : cudeviceget" << std::endl;
    }

  if(cuCtxCreate(&pCtx, 0, dev)  != CUDA_SUCCESS)
    {
      CUERR;
      std::cout << "**** ERROR *** : cuctxcreate" << std::endl;
    }



  return 0;
}

int c_cuda_ctxpopcur(void *contex)
{
  if(cuCtxPopCurrent((CUcontext*)contex) != CUDA_SUCCESS)
    {
      CUERR;
      std::cout << "**** ERROR *** : cuctxcreate" << std::endl;
    }

  return 0;
  }

int c_cuda_ctxpushcur(void *contex)
{
  if(cuCtxPushCurrent(*(CUcontext*)contex) !=CUDA_SUCCESS)
    {
      CUERR;
      std::cout << "**** ERROR *** : cuctxcreate" << std::endl;
    }
 return 0;
 }*/
